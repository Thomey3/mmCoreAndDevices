#include "hip/hip_runtime.h"
#include "Parallel_Computing.cuh"



__global__ void computingimage(uint16_t* in, uint8_t* out, int nSamplesPerPixel, int lineToProcess) {
    uint64_t sum = 0;

    // ʹ�ù����ڴ��������ڴ�����ٶ�
    extern __shared__ uint16_t shared_data[];

    // ���������ݼ��ص������ڴ���
    for (int i = threadIdx.x; i < nSamplesPerPixel * 4; i += blockDim.x) {
        shared_data[i] = in[blockIdx.x * nSamplesPerPixel * 4 + i];
    }
    __syncthreads();

    // �������
    for (int i = 0; i < nSamplesPerPixel; i++) {
        sum += shared_data[i * 4 + threadIdx.x];
    }
    uint16_t average = static_cast<uint16_t>(sum / nSamplesPerPixel);

    // ���Ų�����ֵ�� [0, 255]
    double scaled_value = (average + 8192.0) / 16383.0 * 255.0;
    if (scaled_value < 0.0) scaled_value = 0.0;
    if (scaled_value > 255.0) scaled_value = 255.0;

    out[threadIdx.x * lineToProcess * 512 + blockIdx.x] = static_cast<uint8_t>(scaled_value);
}

int Parallel_Computing(std::vector<uint16_t> data, int nSamplesPerPixel,int linesToProcess, std::vector<std::vector<uint8_t>>& out, int line) {
    // CUDA ��ر���
    uint8_t* d_data2D;
    size_t rows = 4;
    size_t cols = 512 * linesToProcess;
    size_t size2D = rows * cols * sizeof(uint8_t);

    // �� GPU �Ͽ��ٶ�ά uint8_t ����Ŀռ�
    hipError_t err = hipMalloc((void**)&d_data2D, size2D);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc for data2D failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // CUDA ��ر������� data
    uint16_t* d_data;
    size_t sizeData = data.size() * sizeof(uint16_t);

    // �� GPU �Ͽ��� uint16_t ����Ŀռ�
    err = hipMalloc((void**)&d_data, sizeData);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc for data failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // �����ݴ� host ���Ƶ� device
    err = hipMemcpy(d_data, data.data(), sizeData, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy to d_data failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // �����߳̿�������С
    int blockSize = 4;
    int gridSize = 512 * linesToProcess;

    // ���� CUDA �˺���������ͼ�����
    computingimage <<<gridSize, blockSize, nSamplesPerPixel * 4 * sizeof(uint16_t) >> > (d_data, d_data2D, nSamplesPerPixel,linesToProcess);

    // �ȴ� GPU �������
    hipDeviceSynchronize();

    // ����һ�����ڴ洢����Ķ�ά vector
    std::vector<std::vector<uint8_t>> data2D(rows, std::vector<uint8_t>(cols));

    // ������� device ���Ƶ� host
    std::vector<uint8_t> flattened_data2D(rows * cols);
    err = hipMemcpy(flattened_data2D.data(), d_data2D, size2D, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy from d_data2D failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // ����ƽ��������ת��Ϊ��ά��ʽ
    for (size_t i = 0; i < 2; ++i) {
        std::copy(flattened_data2D.begin() + i * cols, flattened_data2D.begin() + (i + 1) * cols, out[i].begin() + line * 512);
    }

    // ����ɺ��ͷ� GPU �ռ�
    err = hipFree(d_data2D);
    if (err != hipSuccess) {
        std::cerr << "CUDA free for data2D failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    err = hipFree(d_data);
    if (err != hipSuccess) {
        std::cerr << "CUDA free for data failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    return 0;
}