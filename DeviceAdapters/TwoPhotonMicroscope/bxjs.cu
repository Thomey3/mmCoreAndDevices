#include "hip/hip_runtime.h"
#include "bxjs.cuh"

__global__ void computingimage(uint16_t* in, uint8_t* out, int nSamplesPerPixel, int lineToProcess) {
    uint64_t sum = 0;

    // �������
    for (int i = 0; i < nSamplesPerPixel; i++) {
        sum += in[i * 4 + blockIdx.x * nSamplesPerPixel * 4 + threadIdx.x];
    }
    uint16_t average = static_cast<uint16_t>(sum / nSamplesPerPixel);

    // ���Ų�����ֵ�� [0, 255]
    double scaled_value = (average + 8192.0) / 16383.0 * 255.0;
    if (scaled_value < 0.0) scaled_value = 0.0;
    if (scaled_value > 255.0) scaled_value = 255.0;

    out[threadIdx.x * lineToProcess * 512 + blockIdx.x] = static_cast<uint8_t>(scaled_value);
}

int bxjs::compute(int width, int line, int processedChannels, int lineToProcess, int nSamplesPerPixel, int nChannels, std::vector<uint16_t> data, std::vector<std::vector<uint8_t>>& out)
{
    if (dataPtr != nullptr) {
        std::memcpy(dataPtr, data.data(), data.size() * sizeof(uint16_t));
    }

    size_t rows = 2;
    size_t cols = 512 * lineToProcess;
    size_t size2D = rows * cols * sizeof(uint8_t);

    // �����߳̿�������С
    int blockSize = 4;
    int gridSize = 512 * lineToProcess;
    computingimage <<<gridSize, blockSize>>> (dataPtr, deviceOutPtr, nSamplesPerPixel,lineToProcess);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        return -4;
    }


    std::vector<uint8_t> flattened_data2D(rows * cols);
    err = hipMemcpy(flattened_data2D.data(), deviceOutPtr, size2D, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        return -5;
    }


    out.resize(2);
    // ����ƽ��������ת��Ϊ��ά��ʽ
    for (size_t i = 0; i < 2; ++i) {
        out[i].resize(cols);
        std::copy(flattened_data2D.begin() + i * cols, flattened_data2D.begin() + (i + 1) * cols, out[i].begin());
    }

    return 0;
}

int bxjs::CreatePageLockedMemory(size_t datasize, size_t imagesize)
{
    // ������ҳ�ڴ�
    hipError_t status = hipHostMalloc((void**)&dataPtr, datasize);
    if (status != hipSuccess) {
        return -1;
    }

    // �����豸�ڴ�
    status = hipMalloc((void**)&deviceOutPtr, imagesize);
    if (status != hipSuccess) {
        hipHostFree(dataPtr);  // ������ʧ�����ͷ��ѷ������ҳ�ڴ�
        return -1;
    }

    //// �����豸�ڴ�
    //status = hipMalloc((void**)&hostOutPtr, imagesize);
    //if (status != hipSuccess) {
    //    hipHostFree(dataPtr);  // ������ʧ�����ͷ��ѷ������ҳ�ڴ�
    //    hipFree(deviceOutPtr);
    //    return -1;
    //}
    ImageSize = imagesize;
    return 0;  // ���� 0 ��ʾ����ɹ�
}

void bxjs::FreePageLockedMemory()
{
    if (dataPtr) {
        hipHostFree(dataPtr);
        dataPtr = nullptr;
    }
    if (deviceOutPtr) {
        hipFree(deviceOutPtr);
        deviceOutPtr = nullptr;
    }
    if (hostOutPtr) {
        hipHostFree(hostOutPtr);
        hostOutPtr = nullptr;
    }
}